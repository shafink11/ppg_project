#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <string>
#include <filesystem>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <NvInfer.h>
#include <NvInferRuntime.h>

#define CUDA_CHECK(status) \
    if (status != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(status) << std::endl; \
        exit(EXIT_FAILURE); \
    }

#define CUDNN_CHECK(status) \
    if (status != HIPDNN_STATUS_SUCCESS) { \
        std::cerr << "cuDNN Error: " << hipdnnGetErrorString(status) << std::endl; \
        exit(EXIT_FAILURE); \
    }

namespace fs = std::filesystem;


std::vector<float> loadWaveform(const std::string& filepath) {
    std::vector<float> waveform;
    std::ifstream file(filepath);
    if (!file.is_open()) {
        std::cerr << "Failed to open file: " << filepath << std::endl;
        return waveform;
    }
    float value;
    while (file >> value) {
        waveform.push_back(value);
    }
    return waveform;
}

std::vector<std::vector<float>> loadAllWaveforms(const std::string& directoryPath) {
    std::vector<std::vector<float>> allWaveforms;
    for (const auto& entry : fs::directory_iterator(directoryPath)) {
        if (entry.is_regular_file()) {
            std::string path = entry.path().string();
            std::vector<float> waveform = loadWaveform(path);
            if (!waveform.empty()) {
                std::cout << "Loaded waveform from " << path 
                          << " with " << waveform.size() << " samples." << std::endl;
                allWaveforms.push_back(waveform);
            }
        }
    }
    return allWaveforms;
}

__global__ void conv1d_kernel(const float* __restrict__ input,
                                const float* __restrict__ kernel,
                                float* __restrict__ output,
                                int input_size,
                                int kernel_size)
{
    extern __shared__ float shared_input[];
    int tid = threadIdx.x;
    int blockStart = blockIdx.x * blockDim.x;
    int out_idx = blockStart + tid;
    for (int i = tid; i < blockDim.x + kernel_size - 1; i += blockDim.x) {
        int input_idx = blockStart + i;
        shared_input[i] = (input_idx < input_size) ? input[input_idx] : 0.0f;
    }
    __syncthreads();
    if (out_idx < (input_size - kernel_size + 1)) {
        float sum = 0.0f;
        #pragma unroll
        for (int k = 0; k < kernel_size; ++k) {
            sum += shared_input[tid + k] * kernel[k];
        }
        output[out_idx] = sum;
    }
}

void runCUDNNConv1D() {
    hipdnnHandle_t cudnn;
    CUDNN_CHECK(hipdnnCreate(&cudnn));

    int batch = 1, channels = 1, height = 1, width = 1024;
    int kernel_width = 5;

    hipdnnTensorDescriptor_t input_desc, output_desc;
    hipdnnFilterDescriptor_t filter_desc;
    hipdnnConvolutionDescriptor_t conv_desc;

    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&input_desc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(input_desc,
                                           HIPDNN_TENSOR_NCHW,
                                           HIPDNN_DATA_FLOAT,
                                           batch, channels, height, width));

    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&filter_desc));
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(filter_desc,
                                           HIPDNN_DATA_FLOAT,
                                           HIPDNN_TENSOR_NCHW,
                                           1, 1, height, kernel_width));

    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&conv_desc));
    int pad_h = 0, pad_w = 0;
    int stride_h = 1, stride_w = 1;
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(conv_desc,
                                                pad_h, pad_w,
                                                stride_h, stride_w,
                                                1, 1,  
                                                HIPDNN_CROSS_CORRELATION,
                                                HIPDNN_DATA_FLOAT));

    int out_n, out_c, out_h, out_w;
    CUDNN_CHECK(hipdnnGetConvolution2dForwardOutputDim(conv_desc,
                                                      input_desc,
                                                      filter_desc,
                                                      &out_n, &out_c, &out_h, &out_w));

    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&output_desc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(output_desc,
                                           HIPDNN_TENSOR_NCHW,
                                           HIPDNN_DATA_FLOAT,
                                           out_n, out_c, out_h, out_w));

    float *d_input, *d_filter, *d_output;
    size_t input_bytes = batch * channels * height * width * sizeof(float);
    size_t filter_bytes = 1 * 1 * height * kernel_width * sizeof(float);
    size_t output_bytes = out_n * out_c * out_h * out_w * sizeof(float);
    CUDA_CHECK(hipMalloc(&d_input, input_bytes));
    CUDA_CHECK(hipMalloc(&d_filter, filter_bytes));
    CUDA_CHECK(hipMalloc(&d_output, output_bytes));

    hipdnnConvolutionFwdAlgo_t algo;
    CUDNN_CHECK(hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                                    input_desc,
                                                    filter_desc,
                                                    conv_desc,
                                                    output_desc,
                                                    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                                    0,
                                                    &algo));

    size_t workspace_bytes = 0;
    CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                        input_desc,
                                                        filter_desc,
                                                        conv_desc,
                                                        output_desc,
                                                        algo,
                                                        &workspace_bytes));
    void* d_workspace = nullptr;
    CUDA_CHECK(hipMalloc(&d_workspace, workspace_bytes));

    const float alpha = 1.0f, beta = 0.0f;
    CUDNN_CHECK(hipdnnConvolutionForward(cudnn,
                                        &alpha,
                                        input_desc,
                                        d_input,
                                        filter_desc,
                                        d_filter,
                                        conv_desc,
                                        algo,
                                        d_workspace,
                                        workspace_bytes,
                                        &beta,
                                        output_desc,
                                        d_output));
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_filter));
    CUDA_CHECK(hipFree(d_output));
    CUDA_CHECK(hipFree(d_workspace));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(input_desc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(output_desc));
    CUDNN_CHECK(hipdnnDestroyFilterDescriptor(filter_desc));
    CUDNN_CHECK(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CUDNN_CHECK(hipdnnDestroy(cudnn));
}

#include <NvInfer.h>
using namespace nvinfer1;

class TRTLogger : public ILogger {
public:
    void log(Severity severity, const char* msg) noexcept override {
        if (severity != Severity::kINFO)
            std::cout << "[TensorRT] " << msg << std::endl;
    }
} gLogger;

ICudaEngine* createTRTInferenceEngine() {
    IBuilder* builder = createInferBuilder(gLogger);
    if (!builder) {
        std::cerr << "Failed to create TensorRT builder!" << std::endl;
        return nullptr;
    }
    INetworkDefinition* network = builder->createNetworkV2(0U);

    ITensor* input = network->addInput("input", DataType::kFLOAT, Dims3{1, 1, 1024});
    if (!input) {
        std::cerr << "Failed to create input tensor!" << std::endl;
        return nullptr;
    }

    Weights {DataType::kFLOAT, std::kload(weights), 0}; 
    IConvolutionLayer* conv = network->addConvolutionNd(*input, 16, DimsHW{1, 5}, emptyWeights, emptyWeights);
    conv->setStrideNd(DimsHW{1, 1});
    conv->setPaddingNd(DimsHW{0, 0});
    if (!conv) {
        std::cerr << "Failed to add convolution layer!" << std::endl;
        return nullptr;
    }

    IActivationLayer* relu = network->addActivation(*conv->getOutput(0), ActivationType::kRELU);
    if (!relu) {
        std::cerr << "Failed to add activation layer!" << std::endl;
        return nullptr;
    }

    network->markOutput(*relu->getOutput(0));

    IBuilderConfig* config = builder->createBuilderConfig();
    config->setMaxWorkspaceSize(1 << 20); 
    ICudaEngine* engine = builder->buildEngineWithConfig(*network, *config);

    network->destroy();
    config->destroy();
    builder->destroy();

    return engine;
}

//------------------------------------------------------------------------------
// Main: Load data and process using CUDA and neural network components
//------------------------------------------------------------------------------
int main() {
    std::string dataDir = "./processed_ppgs";
    std::vector<std::vector<float>> waveforms = loadAllWaveforms(dataDir);
    if (waveforms.empty()) {
        std::cerr << "No waveforms loaded from " << dataDir << std::endl;
        return -1;
    }
    std::vector<float>& ppg = waveforms[0];
    int input_size = ppg.size();
    int kernel_size = 5;
    int output_size = input_size - kernel_size + 1;

    std::vector<float> kernel(kernel_size, 1.0f / kernel_size);

    float* h_input = ppg.data();
    float* h_kernel = kernel.data();
    std::vector<float> h_output(output_size, 0.0f);

    float *d_input, *d_kernel, *d_output;
    CUDA_CHECK(hipMalloc(&d_input, input_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_kernel, kernel_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_output, output_size * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_kernel, h_kernel, kernel_size * sizeof(float), hipMemcpyHostToDevice));

    int threadsPerBlock = 256;
    int blocks = (output_size + threadsPerBlock - 1) / threadsPerBlock;
    size_t sharedMemBytes = (threadsPerBlock + kernel_size - 1) * sizeof(float);
    conv1d_kernel<<<blocks, threadsPerBlock, sharedMemBytes>>>(d_input, d_kernel, d_output, input_size, kernel_size);
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(h_output.data(), d_output, output_size * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < 10 && i < output_size; ++i)
        std::cout << h_output[i] << " ";
    std::cout << std::endl;

    runCUDNNConv1D();

    ICudaEngine* engine = createTRTInferenceEngine();
    if (engine) {
        std::cout << "TensorRT engine created successfully!" << std::endl;
        engine->destroy();
    } else {
        std::cerr << "Failed to create TensorRT engine." << std::endl;
    }

    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_kernel));
    CUDA_CHECK(hipFree(d_output));

    return 0;
}
